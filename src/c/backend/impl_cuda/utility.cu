#include "hip/hip_runtime.h"
#include <cstdint>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <gpu_utility.hpp>

__device__ void gpu_sleep_1_bak(unsigned long sleep_cycles) {
  unsigned long start = clock64();
  volatile unsigned long cycles_elapsed;
  do {
    cycles_elapsed = clock64() - start;
  } while (cycles_elapsed < sleep_cycles);
}

__global__ void gpu_sleep_1(clock_t clock_count) {
  gpu_sleep_1_bak(clock_count);
}

void gpu_busy_sleep(const int device, const unsigned long cycles,
                    uintptr_t stream_ptr) {
  hipStream_t stream = reinterpret_cast<hipStream_t>(stream_ptr);
  gpu_sleep_1<<<1, 1, device, stream>>>(cycles);
}

void event_synchronize(uintptr_t event_ptr) {
  hipEvent_t event = reinterpret_cast<hipEvent_t>(event_ptr);
  hipEventSynchronize(event);
}
void event_wait(uintptr_t event_ptr, uintptr_t stream_ptr) {
  hipEvent_t event = reinterpret_cast<hipEvent_t>(event_ptr);
  hipStream_t stream = reinterpret_cast<hipStream_t>(stream_ptr);

  // The 0 is for the flags.
  // 0 means that the event will be waited on in the default manner.
  // 1 has to do with CUDA graphs.
  hipStreamWaitEvent(stream, event, 0);
};

void stream_synchronize(uintptr_t stream_ptr) {
  hipStream_t stream = reinterpret_cast<hipStream_t>(stream_ptr);
  hipStreamSynchronize(stream);
};

void set_device(int device) { hipSetDevice(device); }

int get_device() {
  int device;
  hipGetDevice(&device);
  return device;
}

int get_num_devices() {
  int num_devices;
  hipGetDeviceCount(&num_devices);
  return num_devices;
}